#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

#define CHECK_CUDA(call) \
    if((call) != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << std::endl; \
        return EXIT_FAILURE; \
    }

#define CHECK_CUSPARSE(call) \
    if((call) != HIPSPARSE_STATUS_SUCCESS) { \
        std::cerr << "cuSPARSE error at " << __FILE__ << ":" << __LINE__ << std::endl; \
        return EXIT_FAILURE; \
    }

#define CHECK_CUBLAS(call) \
    if((call) != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "cuBLAS error at " << __FILE__ << ":" << __LINE__ << std::endl; \
        return EXIT_FAILURE; \
    }

int main() {
    // Initialize cuSPARSE and cuBLAS
    hipsparseHandle_t cusparseHandle;
    hipblasHandle_t cublasHandle;

    CHECK_CUSPARSE(hipsparseCreate(&cusparseHandle));
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    // Your cuSPARSE and cuBLAS code here

    // Clean up
    CHECK_CUSPARSE(hipsparseDestroy(cusparseHandle));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));

    return EXIT_SUCCESS;
}