
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>

__global__ void printNumber(int number)
{
  printf("%d\n", number);
}

int main()
{
  hipStream_t* stream;
  stream = (hipStream_t*)malloc(5 * sizeof(hipStream_t));
  for (int i = 0; i < 5; ++i)
  {
    hipStreamCreate(&stream[i]);
    printNumber<<<1, 1, 0, stream[i]>>>(i);
    hipStreamDestroy(stream[i]);
  }
  hipDeviceSynchronize();
}

