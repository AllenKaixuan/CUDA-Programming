#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>
#include <sys/time.h>

#define DataType double

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len)
{
	//@@ Insert code to implement vector addition here
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < len)
		out[i] = in1[i] + in2[i];
}

double get_time_in_seconds()
{
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + tv.tv_usec / 1000000.0;
}

int main(int argc, char **argv)
{

	int inputLength;
	double start_time, end_time;
	DataType *hostInput1;
	DataType *hostInput2;
	DataType *hostOutput;
	DataType *resultRef;
	DataType *deviceInput1;
	DataType *deviceInput2;
	DataType *deviceOutput;

	//@@ Insert code below to read in inputLength from args
	inputLength = atoi(argv[1]);
	printf("The input length is %d\n", inputLength);

	//@@ Insert code below to read in segment size (S_seg) from args
	int S_seg = 1024; 
	if (argc > 2) {
		S_seg = atoi(argv[2]);
	}
	printf("The segment size is %d\n", S_seg);

	//@@ Insert code below to allocate Host memory for input and output
	size_t size = inputLength * sizeof(DataType);
	hostInput1 = (DataType *)malloc(size);
	hostInput2 = (DataType *)malloc(size);
	hostOutput = (DataType *)malloc(size);
	resultRef  = (DataType *)malloc(size);

	//@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU
	for (int i = 0; i < inputLength; ++i)
	{
		hostInput1[i] = (DataType)rand() / (DataType)RAND_MAX;
		hostInput2[i] = (DataType)rand() / (DataType)RAND_MAX;
		resultRef[i] = hostInput1[i] + hostInput2[i];
	}

	//@@ Insert code below to allocate GPU memory here
	hipMalloc(&deviceInput1, size);
	hipMalloc(&deviceInput2, size);
	hipMalloc(&deviceOutput, size);

	//non-stream
	start_time = get_time_in_seconds();
non-stream
	// Host -> Device
	hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
	hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);

	int threads_per_block = 256;
	int number_of_blocks  = (inputLength + threads_per_block - 1) / threads_per_block;

	vecAdd<<<number_of_blocks, threads_per_block>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);
	hipDeviceSynchronize();

	hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);

	end_time = get_time_in_seconds();
	double nonStreamTime = end_time - start_time;
	printf("Non-stream version total time: %f seconds\n", nonStreamTime);

	// check answer
	double max_error_nonStream = 0;
	for (int i = 0; i < inputLength; ++i)
	{
		double err = fabs(hostOutput[i] - resultRef[i]);
		if (err > max_error_nonStream) {
			max_error_nonStream = err;
		}
	}
	printf("Non-stream version Max Error: %.5f\n", max_error_nonStream);

	//stream version
	//
	//@@ Insert code below to create multiple CUDA streams
	const int NUM_STREAMS = 4;
	hipStream_t streams[NUM_STREAMS];
	for (int i = 0; i < NUM_STREAMS; i++)
	{
		hipStreamCreate(&streams[i]);
	}

	start_time = get_time_in_seconds();

	int totalSegments = (inputLength + S_seg - 1) / S_seg;  

	for (int seg = 0; seg < totalSegments; seg++)
	{
		int streamIdx = seg % NUM_STREAMS;

		int offset = seg * S_seg;
		int len = ((offset + S_seg) < inputLength) ? S_seg : (inputLength - offset);
		size_t segmentSize = len * sizeof(DataType);

		hipMemcpyAsync(deviceInput1 + offset,
		                hostInput1   + offset,
		                segmentSize,
		                hipMemcpyHostToDevice,
		                streams[streamIdx]);

		hipMemcpyAsync(deviceInput2 + offset,
		                hostInput2   + offset,
		                segmentSize,
		                hipMemcpyHostToDevice,
		                streams[streamIdx]);

		// Kernel
		int threads_per_block_stream = 256;
		int number_of_blocks_stream  = (len + threads_per_block_stream - 1) / threads_per_block_stream;

		vecAdd<<<number_of_blocks_stream, threads_per_block_stream, 0, streams[streamIdx]>>>(
		    deviceInput1 + offset,
		    deviceInput2 + offset,
		    deviceOutput + offset,
		    len
		);

		hipMemcpyAsync(hostOutput + offset,
		                deviceOutput + offset,
		                segmentSize,
		                hipMemcpyDeviceToHost,
		                streams[streamIdx]);
	}

	// wait for all streams to be done
	for (int i = 0; i < NUM_STREAMS; i++)
	{
		hipStreamSynchronize(streams[i]);
	}

	end_time = get_time_in_seconds();
	double streamTime = end_time - start_time;
	printf("Streaming version total time (%d segments, %d streams): %f seconds\n", 
	       totalSegments, NUM_STREAMS, streamTime);

	double max_error_stream = 0;
	for (int i = 0; i < inputLength; ++i)
	{
		double err = fabs(hostOutput[i] - resultRef[i]);
		if (err > max_error_stream) {
			max_error_stream = err;
		}
	}
	printf("Streaming version Max Error: %.5f\n", max_error_stream);


	printf("\n--- Performance comparison ---\n");
	printf("Non-stream time      = %f s\n", nonStreamTime);
	printf("Streaming time       = %f s\n", streamTime);
	printf("Speedup (non/stream) = %f\n", nonStreamTime / streamTime);
	printf("------------------------------\n");

	//@@ Insert code below to destroy multiple streams
	for (int i = 0; i < NUM_STREAMS; i++)
	{
		hipStreamDestroy(streams[i]);
	}

	//@@ Free the GPU memory here
	hipFree(deviceInput1);
	hipFree(deviceInput2);
	hipFree(deviceOutput);

	//@@ Free the CPU memory here
	free(hostInput1);
	free(hostInput2);
	free(hostOutput);
	free(resultRef);

	return 0;
}
