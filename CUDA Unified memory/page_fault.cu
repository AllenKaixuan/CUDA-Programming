
#include <hip/hip_runtime.h>
__global__
void deviceKernel(int *a, int N)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < N; i += stride)
  {
    a[i] = 1;
  }
}

void hostFunction(int *a, int N)
{
  for (int i = 0; i < N; ++i)
  {
    a[i] = 1;
  }
}

int main()
{

  int N = 2<<24;
  size_t size = N * sizeof(int);
  int *a;
  hipMallocManaged(&a, size);

  /*
   * Conduct experiments to learn more about the behavior of
   * `cudaMallocManaged`.
   *
   * What happens when unified memory is accessed only by the GPU?  no transfer
   * What happens when unified memory is accessed only by the CPU?  no transfer
   * What happens when unified memory is accessed first by the GPU then the CPU?  device to host
   * What happens when unified memory is accessed first by the CPU then the GPU?  host to device
   *
   * Hypothesize about UM behavior, page faulting specificially, before each
   * experiment, and then verify by running `nsys`.
   */
  hostFunction(a, N);
  deviceKernel<<<256,1024>>>(a, N);
  // cudaDeviceSynchronize();
  

  hipFree(a);
}
